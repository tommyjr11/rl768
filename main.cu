#include "hip/hip_runtime.h"
#include <iostream>
#include "data.h"
#include <vector>
#include <chrono>
int main() {
    
    solVectors d_data_pri;
    solVectors d_data_con;
    // solVectors d_half_uL;
    // solVectors d_half_uR;
    // solVectors d_SLIC_flux;
    checkKernelAttributes();
    allocateDeviceMemory(d_data_pri, d_data_con);
    initDataAndCopyToGPU2(d_data_pri, d_data_con);
    auto start = std::chrono::high_resolution_clock::now();
    double dt = 0.0;
    double t = 0.0;
    int step = 0;
    double tempt = 0.0f;
    double temptt = bubbleR/(sqrt(r*pAir/rhoAir)*Ms);
    std::chrono::duration<double, std::milli> launch_duration;
    // std::cout<<"shack rho: "<< rhoPost<<"shack p: "<< pPost<<"shack u: "<< uPost<<"shack v: "<< vPost<<std::endl;
    for (;;){
        // auto dt_start = std::chrono::high_resolution_clock::now();
        dt = getdtGPU(d_data_pri, 1.4);
        // hipDeviceSynchronize();
        // auto dt_end = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double, std::milli> dt_duration = dt_end - dt_start;
        // std::cout << "dt time: " << dt_duration.count() << " millisecond"<<std::endl;

        // hipDeviceSynchronize();
        std::cout << "step: "<< step << " dt = " << dt <<" t= "<<t<<std::endl;
        step++;
        t = t + dt;
        // auto launch_start = std::chrono::high_resolution_clock::now();
        launchUpdateSLICKernel(d_data_con, dt);
        // auto launch_end = std::chrono::high_resolution_clock::now();
        // launch_duration = launch_duration + (launch_end - launch_start);
        // std::cout << "launch time: " << launch_duration.count() << " millisecond"<<std::endl;


    
        // hipDeviceSynchronize();
        // auto x_half_start = std::chrono::high_resolution_clock::now();
        // computeHalftime(d_data_con,d_half_uL,d_half_uR,dt,1);
        // hipDeviceSynchronize();
        // auto x_half_end = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double, std::milli> x_half_duration = x_half_end - x_half_start;
        // std::cout << "x_half time: " << x_half_duration.count() << " millisecond"<<std::endl;

        // auto x_slic_start = std::chrono::high_resolution_clock::now();
        // computeSLICFlux(d_half_uL,d_half_uR,d_SLIC_flux,dt,1);
        // hipDeviceSynchronize();
        // auto x_slic_end = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double, std::milli> x_slic_duration = x_slic_end - x_slic_start;
        // std::cout << "x_slic time: " << x_slic_duration.count() << " millisecond"<<std::endl;

        // auto x_update_start = std::chrono::high_resolution_clock::now();
        // updateSolution(d_data_con,d_SLIC_flux,dt,1);
        // hipDeviceSynchronize();
        // freeDeviceMemory2(d_half_uL, d_half_uR, d_SLIC_flux);
        // auto x_update_end = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double, std::milli> x_update_duration = x_update_end - x_update_start;
        // std::cout << "x_update time: " << x_update_duration.count() << " millisecond"<<std::endl;

        

        // auto y_half_start = std::chrono::high_resolution_clock::now();
        // computeHalftime(d_data_con,d_half_uL,d_half_uR,dt,2);
        // hipDeviceSynchronize();
        // auto y_half_end = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double, std::milli> y_half_duration = y_half_end - y_half_start;
        // std::cout << "y_half time: " << y_half_duration.count() << " millisecond"<<std::endl;

        // auto y_slic_start = std::chrono::high_resolution_clock::now();
        // computeSLICFlux(d_half_uL,d_half_uR,d_SLIC_flux,dt,2);
        // hipDeviceSynchronize();
        // auto y_slic_end = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double, std::milli> y_slic_duration = y_slic_end - y_slic_start;
        // std::cout << "y_slic time: " << y_slic_duration.count() << " millisecond"<<std::endl;

        // auto y_update_start = std::chrono::high_resolution_clock::now();
        // updateSolution(d_data_con,d_SLIC_flux,dt,2);
        // hipDeviceSynchronize();
        // freeDeviceMemory2(d_half_uL, d_half_uR, d_SLIC_flux);
        // auto y_update_end = std::chrono::high_resolution_clock::now();
        // std::chrono::duration<double, std::milli> y_update_duration = y_update_end - y_update_start;
        // std::cout << "y_update time: " << y_update_duration.count() << " millisecond"<<std::endl;

        applyBoundaryConditions(d_data_con);
        // hipDeviceSynchronize();
        list_con2pri(d_data_con, d_data_pri);
        // hipDeviceSynchronize();
        tempt = t/temptt;
        if (tempt >= t1) break;
        // if (t >= t1) break;
    }
    // std::vector<double> h_rho((nx+4) * (ny+4), 0.0f);
    // std::vector<double> h_vx((nx+4) * (ny+4), 0.0f);
    // std::vector<double> h_vy((nx+4) * (ny+4), 0.0f);
    // std::vector<double> h_p((nx+4) * (ny+4), 0.0f);
    // hipMemcpy(h_rho.data(), d_data_pri.rho, sizeof(double) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
    // hipMemcpy(h_vx.data(), d_data_pri.vx, sizeof(double) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
    // hipMemcpy(h_vy.data(), d_data_pri.vy, sizeof(double) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
    // hipMemcpy(h_p.data(), d_data_pri.p, sizeof(double) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
    // store_data(h_rho, h_vx, h_vy, h_p,dt,1);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Time: " << elapsed.count() << " s\n";
    freeDeviceMemory(d_data_pri, d_data_con);
    return 0;
}