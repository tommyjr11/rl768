#include "hip/hip_runtime.h"
#include <iostream>
#include "data.h"
#include <vector>
#include <chrono>
int main() {
    
    solVectors d_data_pri;
    solVectors d_data_con;
    // solVectors d_half_uL;
    // solVectors d_half_uR;
    // solVectors d_SLIC_flux;
    // checkKernelAttributes();
    allocateDeviceMemory(d_data_pri, d_data_con);
    initDataAndCopyToGPU2(d_data_pri, d_data_con);
    auto start = std::chrono::high_resolution_clock::now();
    double dt = 0.0;
    double t = 0.0;
    int step = 0;
    double tempt = 0.0f;
    double temptt = bubbleR/(sqrt(r*pAir/rhoAir)*Ms);
    // std::cout<<"shack rho: "<< rhoPost<<"shack p: "<< pPost<<"shack u: "<< uPost<<"shack v: "<< vPost<<std::endl;
    for (;;){
        // hipDeviceSynchronize();
        dt = getdtGPU(d_data_pri, 1.4);
        // hipDeviceSynchronize();
        std::cout << "step: "<< step << " dt = " << dt <<" t= "<<t<<std::endl;
        step++;
        t = t + dt;
        launchUpdateSLICKernel(d_data_con, dt);
        // hipDeviceSynchronize();
        // computeHalftime(d_data_con,d_half_uL,d_half_uR,dt,1);
        // hipDeviceSynchronize();
        // computeSLICFlux(d_half_uL,d_half_uR,d_SLIC_flux,dt,1);
        // hipDeviceSynchronize();
        // updateSolution(d_data_con,d_SLIC_flux,dt,1);
        // hipDeviceSynchronize();
        // freeDeviceMemory2(d_half_uL, d_half_uR, d_SLIC_flux);
        // hipDeviceSynchronize();

        // computeHalftime(d_data_con,d_half_uL,d_half_uR,dt,2);
        // hipDeviceSynchronize();
        // computeSLICFlux(d_half_uL,d_half_uR,d_SLIC_flux,dt,2);
        // hipDeviceSynchronize();
        // updateSolution(d_data_con,d_SLIC_flux,dt,2);
        // hipDeviceSynchronize();
        // freeDeviceMemory2(d_half_uL, d_half_uR, d_SLIC_flux);
        // hipDeviceSynchronize();
        applyBoundaryConditions(d_data_con);
        // hipDeviceSynchronize();
        list_con2pri(d_data_con, d_data_pri);
        // hipDeviceSynchronize();
        tempt = t/temptt;
        if (tempt >= t1) break;
        // if (t >= t1) break;
    }
    std::vector<double> h_rho((nx+4) * (ny+4), 0.0f);
    std::vector<double> h_vx((nx+4) * (ny+4), 0.0f);
    std::vector<double> h_vy((nx+4) * (ny+4), 0.0f);
    std::vector<double> h_p((nx+4) * (ny+4), 0.0f);
    hipMemcpy(h_rho.data(), d_data_pri.rho, sizeof(double) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
    hipMemcpy(h_vx.data(), d_data_pri.vx, sizeof(double) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
    hipMemcpy(h_vy.data(), d_data_pri.vy, sizeof(double) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
    hipMemcpy(h_p.data(), d_data_pri.p, sizeof(double) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
    store_data(h_rho, h_vx, h_vy, h_p,dt,1);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;
    std::cout << "Time: " << elapsed.count() << " s\n";
    freeDeviceMemory(d_data_pri, d_data_con);
    return 0;
}