#include "hip/hip_runtime.h"
#include "data.h"
#include "constants.h"  
#include <iostream>
#include <vector>
#include <cmath>       



void allocateDeviceMemory(solVectors &d_data_pri, solVectors &d_data_con) {
    CUDA_CHECK(hipMalloc((void**)&(d_data_pri.rho), (nx+4) * (ny+4) * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&(d_data_pri.vx),  (nx+4) * (ny+4) * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&(d_data_pri.vy),  (nx+4) * (ny+4) * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&(d_data_pri.p),   (nx+4) * (ny+4) * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&(d_data_con.rho), (nx+4) * (ny+4) * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&(d_data_con.vx),  (nx+4) * (ny+4) * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&(d_data_con.vy),  (nx+4) * (ny+4) * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&(d_data_con.p),   (nx+4) * (ny+4) * sizeof(float)));
}

void freeDeviceMemory(solVectors &d_data_pri, solVectors &d_data_con) {
    CUDA_CHECK(hipFree(d_data_pri.rho));
    CUDA_CHECK(hipFree(d_data_pri.vx));
    CUDA_CHECK(hipFree(d_data_pri.vy));
    CUDA_CHECK(hipFree(d_data_pri.p));
    CUDA_CHECK(hipFree(d_data_con.rho));
    CUDA_CHECK(hipFree(d_data_con.vx));
    CUDA_CHECK(hipFree(d_data_con.vy));
    CUDA_CHECK(hipFree(d_data_con.p));
}

__device__ void get_con(const float *pri, float *con)
{
    con[0] = pri[0];
    con[1] = pri[0]*pri[1];
    con[2] = pri[0]*pri[2];
    con[3] = 0.5*pri[0]*(pow(pri[1],2)+pow(pri[2],2))+pri[3]/(1.4-1);
}

__device__ void get_pri(const float *con, float *pri)
{
    pri[0] = con[0];
    pri[1] = con[1]/con[0];
    pri[2] = con[2]/con[0];
    pri[3] = (1.4-1)*(con[3]-0.5*con[0]*(pow(pri[1],2)+pow(pri[2],2)));
}

__global__ void kernel_pri2con(const solVectors d_data_pri, solVectors d_data_con, 
    int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx+4 && j < ny+4) {
    int idx = j * (nx+4) + i;
    float pri[4];
    pri[0] = d_data_pri.rho[idx];  // rho
    pri[1] = d_data_pri.vx [idx];  // vx
    pri[2] = d_data_pri.vy [idx];  // vy
    pri[3] = d_data_pri.p  [idx];  // p
    float con[4];
    get_con(pri,con);
    d_data_con.rho[idx] = con[0];  // rho
    d_data_con.vx [idx] = con[1];  // rho*vx
    d_data_con.vy [idx] = con[2];  // rho*vy
    d_data_con.p  [idx] = con[3];  // E (总能量)
    }
}

__global__ void kernel_con2pri(const solVectors d_data_con, solVectors d_data_pri, 
    int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx+4 && j < ny+4) {
    int idx = j * (nx+4) + i;
    float con[4];
    con[0] = d_data_con.rho[idx];  // rho
    con[1] = d_data_con.vx [idx];  // rho*vx
    con[2] = d_data_con.vy [idx];  // rho*vy
    con[3] = d_data_con.p  [idx];  // E (总能量)
    float pri[4];
    get_pri(con,pri);
    d_data_pri.rho[idx] = pri[0];  // rho
    d_data_pri.vx [idx] = pri[1];  // vx
    d_data_pri.vy [idx] = pri[2];  // vy
    d_data_pri.p  [idx] = pri[3];  // p
    }
}

void initDataAndCopyToGPU(solVectors &d_data_pri,solVectors d_data_con)
{
    std::vector<float> h_rho((nx+4) * (ny+4), 0.0f);
    std::vector<float> h_vx ((nx+4) * (ny+4), 0.0f);
    std::vector<float> h_vy ((nx+4) * (ny+4), 0.0f);
    std::vector<float> h_p  ((nx+4) * (ny+4), 0.0f);

    // 初始化
    for (int j = 0; j < ny+4; j++) {
        for (int i = 0; i < nx+4; i++) {
            int idx = j * (nx+4) + i;

            // 将(i,j)映射到物理坐标 (x, y)
            float x = (i - ghost + 0.5f) * dx; 
            float y = (j - ghost + 0.5f) * dy;

            // 根据坐标区域，给出不同初值（示例）
            if (x < 0.5f) {
                if (y < 0.5f) {
                    h_rho[idx] = 0.138f;
                    h_vx [idx] = 1.206f;
                    h_vy [idx] = 1.206f;
                    h_p  [idx] = 0.029f;
                } else {
                    h_rho[idx] = 0.5323f;
                    h_vx [idx] = 1.206f;
                    h_vy [idx] = 0.0f;
                    h_p  [idx] = 0.3f;
                }
            } else {
                if (y < 0.5f) {
                    h_rho[idx] = 0.5323f;
                    h_vx [idx] = 0.0f;
                    h_vy [idx] = 1.206f;
                    h_p  [idx] = 0.3f;
                } else {
                    h_rho[idx] = 1.5f;
                    h_vx [idx] = 0.0f;
                    h_vy [idx] = 0.0f;
                    h_p  [idx] = 1.5f;
                }
            }
        }
    }

    // 拷贝到 GPU
    size_t sizeBytes = (nx+4) * (ny+4) * sizeof(float);
    CUDA_CHECK(hipMemcpy(d_data_pri.rho, h_rho.data(), sizeBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data_pri.vx,  h_vx.data(),  sizeBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data_pri.vy,  h_vy.data(),  sizeBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data_pri.p,   h_p.data(),   sizeBytes, hipMemcpyHostToDevice));
    // 将 d_data_pri 的数据拷贝到 d_data_con
    dim3 blockSize(16, 16);
    dim3 gridSize((nx+4+15)/16, (ny+4+15)/16);
    kernel_pri2con<<<gridSize, blockSize>>>(d_data_pri, d_data_con, nx, ny);
    hipDeviceSynchronize();
}

__global__ void getMaxSpeedKernel(
    const float* __restrict__ rho,
    const float* __restrict__ vx,
    const float* __restrict__ vy,
    const float* __restrict__ p,
    float* __restrict__ blockMax,
    int totalSize,
    float r)
{
    extern __shared__ float sdata[];

    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    int tid = threadIdx.x;

    float localMax = 0.0f;

    // **优化1: grid-stride 循环**
    for (int idx = globalThreadId; idx < totalSize; idx += step) {
        float c   = sqrtf(r * __ldg(&p[idx]) / __ldg(&rho[idx]));  // **优化2: __ldg() 提高访存效率**
        float spx = fabsf(__ldg(&vx[idx])) + c;
        float spy = fabsf(__ldg(&vy[idx])) + c;
        localMax  = fmaxf(localMax, fmaxf(spx, spy));
    }

    // **优化3: Warp-level reduction**
    sdata[tid] = localMax;
    __syncthreads();

    // **Warp 级别归约**
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        localMax = fmaxf(localMax, __shfl_down_sync(0xffffffff, localMax, offset));
    }

    // **优化4: 使用 warp shuffle 归约到 warp 0**
    if ((tid % warpSize) == 0) {
        sdata[tid / warpSize] = localMax;
    }
    __syncthreads();

    // **仅 block 内 thread 0 进行最终归约**
    if (tid == 0) {
        for (int i = 1; i < blockDim.x / warpSize; i++) {
            localMax = fmaxf(localMax, sdata[i]);
        }
        blockMax[blockIdx.x] = localMax;
    }
}


float getmaxspeedGPU(const solVectors &d_data_pri, float r)
{
    // 这里简化一下，直接把 totalSize = (nx+4)*(ny+4)
    int totalSize = (nx+4) * (ny+4);

    int blockSize = 64;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;

    float *d_blockMax = nullptr;
    CUDA_CHECK(hipMalloc(&d_blockMax, gridSize * sizeof(float)));

    int sharedMemSize = blockSize * sizeof(float);
    
    getMaxSpeedKernel<<<gridSize, blockSize, sharedMemSize>>>(
        d_data_pri.rho,
        d_data_pri.vx,
        d_data_pri.vy,
        d_data_pri.p,
        d_blockMax,
        totalSize,
        r
    );
    CUDA_CHECK(hipDeviceSynchronize());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Kernel Error: " << hipGetErrorString(err) << std::endl;
    }
    std::vector<float> h_blockMax(gridSize, 0.0f);
    CUDA_CHECK(hipMemcpy(h_blockMax.data(), d_blockMax, gridSize * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_blockMax));

    float maxSpeed = 0.0f;
    for (int i = 0; i < gridSize; i++) {
        maxSpeed = fmaxf(maxSpeed, h_blockMax[i]);
    }

    return maxSpeed;
}

float getdtGPU(const solVectors &d_data_pri, float r)
{
    float maxSpeed = getmaxspeedGPU(d_data_pri, r);
    if (maxSpeed < 1e-15f) {
        return 1.0e10f; // 给一个很大的dt
    }
    // 选一个最小网格尺度
    float minDxDy = fminf(dx, dy);
    float dt = C * minDxDy / maxSpeed;
    return dt;
}

// 内核函数：更新左右边界
__global__ void boundary_left_right(solVectors u, int truenx, int trueny) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < trueny) {
        int rowStart = i * truenx;
        // 左边界：将第0列和第1列赋值为第2列的值
        u.p[rowStart + 0] = u.p[rowStart + 2];
        u.p[rowStart + 1] = u.p[rowStart + 2];
        u.rho[rowStart + 0] = u.rho[rowStart + 2];
        u.rho[rowStart + 1] = u.rho[rowStart + 2];
        u.vx[rowStart + 0] = u.vx[rowStart + 2];
        u.vx[rowStart + 1] = u.vx[rowStart + 2];
        u.vy[rowStart + 0] = u.vy[rowStart + 2];
        u.vy[rowStart + 1] = u.vy[rowStart + 2];
        
        // 右边界：将倒数第1列和倒数第2列赋值为倒数第3列的值
        u.p[rowStart + (truenx - 2)] = u.p[rowStart + (truenx - 3)];
        u.p[rowStart + (trueny - 1)] = u.p[rowStart + (trueny - 3)];
        u.rho[rowStart + (truenx - 2)] = u.rho[rowStart + (truenx - 3)];
        u.rho[rowStart + (trueny - 1)] = u.rho[rowStart + (trueny - 3)];
        u.vx[rowStart + (truenx - 2)] = u.vx[rowStart + (truenx - 3)];
        u.vx[rowStart + (trueny - 1)] = u.vx[rowStart + (trueny - 3)];
        u.vy[rowStart + (truenx - 2)] = u.vy[rowStart + (truenx - 3)];
        u.vy[rowStart + (trueny - 1)] = u.vy[rowStart + (trueny - 3)];
    }
}

// 内核函数：更新上下边界
__global__ void boundary_top_bottom(solVectors u, int truenx, int trueny) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < truenx) {
        // 上边界：将第0行和第1行赋值为第2行的值
        u.p[0 * truenx + j] = u.p[2 * truenx + j];
        u.p[1 * truenx + j] = u.p[2 * truenx + j];
        u.rho[0 * truenx + j] = u.rho[2 * truenx + j];
        u.rho[1 * truenx + j] = u.rho[2 * truenx + j];
        u.vx[0 * truenx + j] = u.vx[2 * truenx + j];
        u.vx[1 * truenx + j] = u.vx[2 * truenx + j];
        u.vy[0 * truenx + j] = u.vy[2 * truenx + j];
        u.vy[1 * truenx + j] = u.vy[2 * truenx + j];
        // 下边界：将倒数第1行和倒数第2行赋值为倒数第3行的值
        u.p[(trueny - 2) * truenx + j] = u.p[(trueny - 3) * truenx + j];
        u.p[(trueny - 1) * truenx + j] = u.p[(trueny - 3) * truenx + j];
        u.rho[(trueny - 2) * truenx + j] = u.rho[(trueny - 3) * truenx + j];
        u.rho[(trueny - 1) * truenx + j] = u.rho[(trueny - 3) * truenx + j];
        u.vx[(trueny - 2) * truenx + j] = u.vx[(trueny - 3) * truenx + j];
        u.vx[(trueny - 1) * truenx + j] = u.vx[(trueny - 3) * truenx + j];
    }
}

// 边界条件更新函数：接收指向GPU内存的指针
void applyBoundaryConditions(solVectors &d_u) {
    int threadsPerBlock = 128;
    int truenx = nx + 4;
    int trueny = ny + 4;
    // 更新左右边界：每个线程处理一行
    int blocksLR = ((ny+4) + threadsPerBlock - 1) / threadsPerBlock;
    boundary_left_right<<<blocksLR, threadsPerBlock>>>(d_u, truenx, trueny);
    // 更新上下边界：每个线程处理一列
    int blocksTB = (nx + threadsPerBlock - 1) / threadsPerBlock;
    boundary_top_bottom<<<blocksTB, threadsPerBlock>>>(d_u, truenx, trueny);
    // 等待内核执行完成
    hipDeviceSynchronize();
}



__device__ float limiterL2(float smaller, float larger) {
    if (larger == 0.0)
        return (smaller == 0.0) ? 0.0 : 1.0;
    float R = smaller / larger;
    return fminf(fmaxf(R, 0.0), 1.0);
}

__device__ float limiterR2(float smaller, float larger) {
    if (larger == 0.0)
        return 0.0;
        float R = smaller / larger;
    return (R <= 0.0) ? 0.0 : ((R <= 1.0) ? R : fmin(1.0, 2.0/(1.0+R)));
}

__device__ void get_flux_x(const float *pri, float *flux) {
    flux[0] = pri[0]*pri[1];
    flux[1] = pri[0]*pri[1]*pri[1] + pri[3];
    flux[2] = pri[0]*pri[1]*pri[2];
    float Energy = 0.5*pri[0]*(pri[1]*pri[1] + pri[2]*pri[2]) + pri[3]/(1.4-1.0);
    flux[3] = pri[1]*(pri[3] + Energy);
}

__device__ void get_flux_y(const float *pri, float *flux) {
    flux[0] = pri[0]*pri[2];
    flux[1] = pri[0]*pri[1]*pri[2];
    flux[2] = pri[0]*pri[2]*pri[2] + pri[3];
    float Energy = 0.5*pri[0]*(pri[1]*pri[1] + pri[2]*pri[2]) + pri[3]/(1.4-1.0);
    flux[3] = pri[2]*(pri[3] + Energy);
}

__global__ void computeHalftimeKernel_x(
    const solVectors &d_data_con,
    solVectors d_half_uL,
    solVectors d_half_uR, 
    float dt,
    float dx,
    int nx, int ny
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= (nx+3) || j >= (ny+4)) {
        return;
    }
    int stride = (nx + 4);
    int idx      = j*stride + i;
    int idx_left = j*stride + (i - 1);
    int idx_right= j*stride + (i + 1);

    // --- Step 1: 读取 con(i,j), con(i-1,j), con(i+1,j) ---
    float conM[4];  // con(i,j)
    float conL[4];  // con(i-1,j)
    float conR[4];  // con(i+1,j)
    conM[0] = d_data_con.rho[idx];
    conM[1] = d_data_con.vx [idx];  // 这里 vx 里实际存的是 rho*u
    conM[2] = d_data_con.vy [idx];  // 这里 vy 里实际存的是 rho*v
    conM[3] = d_data_con.p  [idx];  // E (总能量)

    conL[0] = d_data_con.rho[idx_left];
    conL[1] = d_data_con.vx [idx_left];
    conL[2] = d_data_con.vy [idx_left];
    conL[3] = d_data_con.p  [idx_left];

    conR[0] = d_data_con.rho[idx_right];
    conR[1] = d_data_con.vx [idx_right];
    conR[2] = d_data_con.vy [idx_right];
    conR[3] = d_data_con.p  [idx_right];

    // --- Step 2: 斜率限制，得到 tempL, tempR (仍在保守量空间) ---
    float tempL[4], tempR[4];
    for (int k = 0; k < 4; k++) {
        float temp1 = conM[k] - conL[k];  // i - (i-1)
        float temp2 = conR[k] - conM[k];  // (i+1) - i
        float di = 0.5f * (temp1 + temp2);

        // 这里分别调用 limiterL2 / limiterR2：
        float phiL = limiterL2(temp1, temp2);
        float phiR = limiterR2(temp1, temp2);

        // 得到左右临时状态
        tempL[k] = conM[k] - 0.5f * di * phiL;
        tempR[k] = conM[k] + 0.5f * di * phiR;
    }
    // --- Step 3: 将 tempL, tempR 转为原始量 priL, priR，并计算通量 fluxL, fluxR ---
    float priL[4], priR[4];
    get_pri(tempL, priL);
    get_pri(tempR, priR);

    float fluxL[4], fluxR[4];
    get_flux_x(priL, fluxL);
    get_flux_x(priR, fluxR);

    // --- Step 4: 半步更新 (回到保守量空间) ---
    // tempL, tempR 各减去 0.5*(dt/dx)*(fluxR - fluxL)
    for (int k = 0; k < 4; k++) {
        float delta = 0.5f * (dt / dx) * (fluxR[k] - fluxL[k]);
        tempL[k] = tempL[k] - delta;
        tempR[k] = tempR[k] - delta;
    }

    // --- Step 5: 把结果存到 half_uL, half_uR 里 ---
    // half_uL, half_uR 大小可能是 (nx-2)*ny
    // 所以对应的一维索引 out_idx = out_j*(nx-2) + out_i
    // 这里 out_j = j (0 <= j < ny)
    if (j >= 0 && (i-1) < (nx - 2)) {
        int out_idx = j*(nx - 2) + (i-1);
        // 写入 half_uL
        d_half_uL.rho[out_idx] = tempL[0];
        d_half_uL.vx [out_idx] = tempL[1];
        d_half_uL.vy [out_idx] = tempL[2];
        d_half_uL.p  [out_idx] = tempL[3];

        // 写入 half_uR
        d_half_uR.rho[out_idx] = tempR[0];
        d_half_uR.vx [out_idx] = tempR[1];
        d_half_uR.vy [out_idx] = tempR[2];
        d_half_uR.p  [out_idx] = tempR[3];
    }
}

void computeHalftime(
    const solVectors &d_data_con,
    solVectors &d_half_uL,
    solVectors &d_half_uR,
    float dt,
    int choice
)
{
    if (choice == 1)
    {
        CUDA_CHECK(hipMalloc((void**)&(d_half_uL.rho), (nx+2) * (ny+4) * sizeof(float)));
        CUDA_CHECK(hipMalloc((void**)&(d_half_uL.vx),  (nx+2) * (ny+4) * sizeof(float)));
        CUDA_CHECK(hipMalloc((void**)&(d_half_uL.vy),  (nx+2) * (ny+4) * sizeof(float)));
        CUDA_CHECK(hipMalloc((void**)&(d_half_uL.p),   (nx+2) * (ny+4) * sizeof(float)));
        CUDA_CHECK(hipMalloc((void**)&(d_half_uR.rho), (nx+2) * (ny+4) * sizeof(float)));
        CUDA_CHECK(hipMalloc((void**)&(d_half_uR.vx),  (nx+2) * (ny+4) * sizeof(float)));
        CUDA_CHECK(hipMalloc((void**)&(d_half_uR.vy),  (nx+2) * (ny+4) * sizeof(float)));
        CUDA_CHECK(hipMalloc((void**)&(d_half_uR.p),   (nx+2) * (ny+4) * sizeof(float)));
        dim3 block(16, 16);
        dim3 grid( (nx+block.x-1)/block.x, (ny+block.y-1)/block.y );
        computeHalftimeKernel_x<<<grid, block>>>(
            d_data_con,    
            d_half_uL,     
            d_half_uR,     
            dt, dx, 
            nx, ny
        );
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA kernel launch failed: " << hipGetErrorString(err) << std::endl;
            exit(-1);
        }
    }
}

