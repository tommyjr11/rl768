#include "hip/hip_runtime.h"
#include "data.h"
#include "constants.h"  
#include <iostream>
#include <vector>
#include <cmath>       



void allocateDeviceMemory(solVectors &d_data) {
    CUDA_CHECK(hipMalloc((void**)&(d_data.rho), (nx+4) * (ny+4) * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&(d_data.vx),  (nx+4) * (ny+4) * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&(d_data.vy),  (nx+4) * (ny+4) * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&(d_data.p),   (nx+4) * (ny+4) * sizeof(float)));
}

void freeDeviceMemory(solVectors &d_data) {
    CUDA_CHECK(hipFree(d_data.rho));
    CUDA_CHECK(hipFree(d_data.vx));
    CUDA_CHECK(hipFree(d_data.vy));
    CUDA_CHECK(hipFree(d_data.p));
}

void initDataAndCopyToGPU(solVectors &d_data)
{
    std::vector<float> h_rho((nx+4) * (ny+4), 0.0f);
    std::vector<float> h_vx ((nx+4) * (ny+4), 0.0f);
    std::vector<float> h_vy ((nx+4) * (ny+4), 0.0f);
    std::vector<float> h_p  ((nx+4) * (ny+4), 0.0f);

    // 初始化
    for (int j = 0; j < ny+4; j++) {
        for (int i = 0; i < nx+4; i++) {
            int idx = j * (nx+4) + i;

            // 将(i,j)映射到物理坐标 (x, y)
            float x = (i - ghost + 0.5f) * dx; 
            float y = (j - ghost + 0.5f) * dy;

            // 根据坐标区域，给出不同初值（示例）
            if (x < 0.5f) {
                if (y < 0.5f) {
                    h_rho[idx] = 0.138f;
                    h_vx [idx] = 1.206f;
                    h_vy [idx] = 1.206f;
                    h_p  [idx] = 0.029f;
                } else {
                    h_rho[idx] = 0.5323f;
                    h_vx [idx] = 1.206f;
                    h_vy [idx] = 0.0f;
                    h_p  [idx] = 0.3f;
                }
            } else {
                if (y < 0.5f) {
                    h_rho[idx] = 0.5323f;
                    h_vx [idx] = 0.0f;
                    h_vy [idx] = 1.206f;
                    h_p  [idx] = 0.3f;
                } else {
                    h_rho[idx] = 1.5f;
                    h_vx [idx] = 0.0f;
                    h_vy [idx] = 0.0f;
                    h_p  [idx] = 1.5f;
                }
            }
        }
    }

    // 拷贝到 GPU
    size_t sizeBytes = (nx+4) * (ny+4) * sizeof(float);
    CUDA_CHECK(hipMemcpy(d_data.rho, h_rho.data(), sizeBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data.vx,  h_vx.data(),  sizeBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data.vy,  h_vy.data(),  sizeBytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data.p,   h_p.data(),   sizeBytes, hipMemcpyHostToDevice));
}

__global__ void getMaxSpeedKernel(
    const float* __restrict__ rho,
    const float* __restrict__ vx,
    const float* __restrict__ vy,
    const float* __restrict__ p,
    float* __restrict__ blockMax,
    int totalSize,
    float r)
{
    extern __shared__ float sdata[];

    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    int tid = threadIdx.x;

    float localMax = 0.0f;

    // **优化1: grid-stride 循环**
    for (int idx = globalThreadId; idx < totalSize; idx += step) {
        float c   = sqrtf(r * __ldg(&p[idx]) / __ldg(&rho[idx]));  // **优化2: __ldg() 提高访存效率**
        float spx = fabsf(__ldg(&vx[idx])) + c;
        float spy = fabsf(__ldg(&vy[idx])) + c;
        localMax  = fmaxf(localMax, fmaxf(spx, spy));
    }

    // **优化3: Warp-level reduction**
    sdata[tid] = localMax;
    __syncthreads();

    // **Warp 级别归约**
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        localMax = fmaxf(localMax, __shfl_down_sync(0xffffffff, localMax, offset));
    }

    // **优化4: 使用 warp shuffle 归约到 warp 0**
    if ((tid % warpSize) == 0) {
        sdata[tid / warpSize] = localMax;
    }
    __syncthreads();

    // **仅 block 内 thread 0 进行最终归约**
    if (tid == 0) {
        for (int i = 1; i < blockDim.x / warpSize; i++) {
            localMax = fmaxf(localMax, sdata[i]);
        }
        blockMax[blockIdx.x] = localMax;
    }
}


float getmaxspeedGPU(const solVectors &d_data, float r)
{
    // 这里简化一下，直接把 totalSize = (nx+4)*(ny+4)
    int totalSize = (nx+4) * (ny+4);

    int blockSize = 64;
    int gridSize  = (totalSize + blockSize - 1) / blockSize;

    float *d_blockMax = nullptr;
    CUDA_CHECK(hipMalloc(&d_blockMax, gridSize * sizeof(float)));

    int sharedMemSize = blockSize * sizeof(float);
    
    getMaxSpeedKernel<<<gridSize, blockSize, sharedMemSize>>>(
        d_data.rho,
        d_data.vx,
        d_data.vy,
        d_data.p,
        d_blockMax,
        totalSize,
        r
    );
    CUDA_CHECK(hipDeviceSynchronize());
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Kernel Error: " << hipGetErrorString(err) << std::endl;
    }
    std::vector<float> h_blockMax(gridSize, 0.0f);
    CUDA_CHECK(hipMemcpy(h_blockMax.data(), d_blockMax, gridSize * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_blockMax));

    float maxSpeed = 0.0f;
    for (int i = 0; i < gridSize; i++) {
        maxSpeed = fmaxf(maxSpeed, h_blockMax[i]);
    }

    return maxSpeed;
}

float getdtGPU(const solVectors &d_data, float r)
{
    float maxSpeed = getmaxspeedGPU(d_data, r);
    std::cout<<"maxSpeed: "<<maxSpeed<<std::endl;

    // 避免除以0
    if (maxSpeed < 1e-15f) {
        return 1.0e10f; // 给一个很大的dt
    }

    // 选一个最小网格尺度
    float minDxDy = fminf(dx, dy);
    float dt = C * minDxDy / maxSpeed;
    return dt;
}