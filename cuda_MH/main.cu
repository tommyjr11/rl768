#include <iostream>
#include "constants.h"
#include "data.h"
#include <vector>

int main() {
    solVectors d_data_pri;
    solVectors d_data_con;
    solVectors d_half_uL;
    solVectors d_half_uR;
    solVectors d_SLIC_flux;
    allocateDeviceMemory(d_data_pri, d_data_con);
    initDataAndCopyToGPU(d_data_pri, d_data_con);
    float dt = 0.0f;
    std::vector<float> h_rho((nx+4) * (ny+4), 0.0f);
    std::vector<float> h_vx((nx+4) * (ny+4), 0.0f);
    std::vector<float> h_vy((nx+4) * (ny+4), 0.0f);
    std::vector<float> h_p((nx+4) * (ny+4), 0.0f);
    float t = 0.0f;
    int step = 0;
    for (;;){
        dt = getdtGPU(d_data_pri, 1.4f);
        std::cout << "step: "<< step << " dt = " << dt << std::endl;
        step++;
        if (t >= t1) break;
        t = t + dt;
        applyBoundaryConditions(d_data_con);
        // x 方向
        computeHalftime(d_data_con,d_half_uL,d_half_uR,dt,1);
        computeSLICFlux(d_half_uL,d_half_uR,d_SLIC_flux,dt,1);
        updateSolution(d_data_con,d_SLIC_flux,dt,1);
        freeDeviceMemory2(d_half_uL, d_half_uR, d_SLIC_flux);

        computeHalftime(d_data_con,d_half_uL,d_half_uR,dt,2);
        computeSLICFlux(d_half_uL,d_half_uR,d_SLIC_flux,dt,2);
        updateSolution(d_data_con,d_SLIC_flux,dt,2);
        freeDeviceMemory2(d_half_uL, d_half_uR, d_SLIC_flux);
        list_con2pri(d_data_con, d_data_pri);

        // 取出 d_data_pri
        hipMemcpy(h_rho.data(), d_data_pri.rho, sizeof(float) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
        hipMemcpy(h_vx.data(), d_data_pri.vx, sizeof(float) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
        hipMemcpy(h_vy.data(), d_data_pri.vy, sizeof(float) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
        hipMemcpy(h_p.data(), d_data_pri.p, sizeof(float) * (nx+4) * (ny+4), hipMemcpyDeviceToHost);
        store_data(h_rho, h_vx, h_vy, h_p,dt,step);

    }
    freeDeviceMemory(d_data_pri, d_data_con);
    return 0;
}